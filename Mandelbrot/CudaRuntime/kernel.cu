﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
// --------------------------------------------------------------------
// Dll Exports
// --------------------------------------------------------------------
extern "C" __declspec(dllexport)
hipError_t setCudaDevice(int device);

extern "C" __declspec(dllexport)
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

extern "C" __declspec(dllexport)
int computeMandelWithCuda(int* output, int width, int height,
double centerX, double centerY, double mandelWidth, double mandelHeight, int maxDepth);

// --------------------------------------------------------------------
// CUDA Kernels
// --------------------------------------------------------------------
// Accepts pointers to three arrays and calculates c = a + b.
__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

// CUDA kernel for computing Mandelbrot iteration counts
__global__ void mandelKernel(int* output, int width, int height, double centerX, double centerY, double mandelWidth, double mandelHeight, int maxDepth)
{
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (column >= width || row >= height) return;
    
	// Convert pixel coordinates to Mandelbrot rectangle coordinates
    double cx = centerX - mandelWidth + column * ((mandelWidth * 2.0) / width);
	double cy = centerY - mandelHeight + row * ((mandelHeight * 2.0) / height);
    
    // Compute iteration count using escape-time algorithm
    int result = 0;
    double x = 0.0;
    double y = 0.0;
    double xx = 0.0, yy = 0.0;
    
    while (xx + yy <= 4.0 && result < maxDepth) {
        xx = x * x;
   yy = y * y;
        double xtmp = xx - yy + cx;
        y = 2.0 * x * y + cy;
        x = xtmp;
        result++;
    }
    
    output[row * width + column] = result;
}

// --------------------------------------------------------------------
// Main Function
// --------------------------------------------------------------------
// The main() function creates three arrays, calls addWithCuda(),
// and prints out the result. Finally, it resets the CUDA device (GPU).
int main()
{
	hipError_t cudaStatus = hipSuccess;

	// Create three (stack-allocated) vectors.
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Set CUDA device (GPU).
	cudaStatus = setCudaDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "setCudaDevice failed!");
		return 1;
	}

	// Add vectors in parallel.
	cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// Print out the result.
	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", c[0], c[1], c[2], c[3], c[4]);

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}
// --------------------------------------------------------------------
// Helper Functions
// --------------------------------------------------------------------
// This function accepts a CUDA device ID, and sets the CUDA device (GPU).
hipError_t setCudaDevice(int device)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	// Note! Can be omitted if the default device (0) is used.
	cudaStatus = hipSetDevice(device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
	}
	return cudaStatus;
}
// This function uses CUDA to add vectors in parallel.
// It accepts pointers to three arrays, sets the CUDA device (GPU),
// allocates device buffers and copied the host buffers to them,
// launches a vector addition CUDA kernel, copied the device output
// buffer to the host output buffer (array), and finally
// frees the device buffers.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (two input, one output) .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
			goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr,
			"hipDeviceSynchronize returned error code %d after launching addKernel!\n",
			cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
		return cudaStatus;
}

// This function uses CUDA to compute Mandelbrot iteration counts in parallel.
// It allocates device buffer, copies parameters, launches the Mandelbrot kernel,
// and copies results back to host.
int computeMandelWithCuda(int* output, int width, int height,
    double centerX, double centerY, double mandelWidth, double mandelHeight, int maxDepth)
{
    int* dev_output = 0;
    hipError_t cudaStatus;
    
    // Allocate GPU buffer for output
    int totalPixels = width * height;
    cudaStatus = hipMalloc((void**)&dev_output, totalPixels * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
		return 1;
	}
    
    // Initialize output buffer to zero
    cudaStatus = hipMemset(dev_output, 0, totalPixels * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemset failed!");
        hipFree(dev_output);
		return 1;
    }
    
    // Launch kernel with 2D grid
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    mandelKernel<<<numBlocks, threadsPerBlock>>>(dev_output, width, height, centerX, centerY, mandelWidth, mandelHeight, maxDepth);
    
    // Check for kernel launch errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "mandelKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_output);
        return 1;
    }
    
    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mandelKernel!\n", cudaStatus);
        hipFree(dev_output);
        return 1;
    }
    
    // Copy output from GPU to host
    cudaStatus = hipMemcpy(output, dev_output, totalPixels * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_output);
        return 1;
    }
    
    hipFree(dev_output);
    return 0; // Success
}
